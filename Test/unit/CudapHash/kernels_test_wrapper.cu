#include "kernels.cuh"
#include "kernels_test_wrapper.h"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

// Map test structures to actual kernel structures
struct GpuData {
    unsigned char* decodedPtr = nullptr;
    float* resizedPtr = nullptr;
    size_t originalWidth;
    size_t originalHeight;
};

struct alignas(16) pHash {
    uint64_t words[2] = { 0, 0 };
};

// Helper function to copy TestGpuData to GpuData
GpuData convertToGpuData(const TestGpuData& testData) {
    GpuData data;
    data.decodedPtr = testData.decodedPtr;
    data.resizedPtr = testData.resizedPtr;
    data.originalWidth = testData.originalWidth;
    data.originalHeight = testData.originalHeight;
    return data;
}

// Wrapper function for testing dctMatrixKernel
void testDctMatrixKernel(float* h_T, int N, float invSqrtN, float sqrtTwoDivN) {
    float* d_T;

    hipMalloc(&d_T, N * N * sizeof(float));
    hipMemcpy(d_T, h_T, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks(N);
    dim3 threads(N);
    dctMatrixKernel<<<blocks, threads>>>(d_T, invSqrtN, sqrtTwoDivN);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_T);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_T, d_T, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_T);
}

// Wrapper function for testing batchBicubicResizeKernel
void testBatchBicubicResizeKernel(
    const TestGpuData* h_gpuDataArray,
    int batchSize,
    size_t outSize,
    const unsigned char** h_inputImages,
    float** h_outputImages)
{
    GpuData* d_gpuDataArray;
    hipMalloc(&d_gpuDataArray, batchSize * sizeof(GpuData));

    std::vector<GpuData> tempGpuData(batchSize);
    std::vector<unsigned char*> d_inputPtrs(batchSize);
    std::vector<float*> d_outputPtrs(batchSize);

    for (int i = 0; i < batchSize; i++) {
        size_t inputSize = h_gpuDataArray[i].originalWidth * h_gpuDataArray[i].originalHeight;
        hipMalloc(&d_inputPtrs[i], inputSize * sizeof(unsigned char));
        hipMalloc(&d_outputPtrs[i], outSize * outSize * sizeof(float));

        hipMemcpy(d_inputPtrs[i], h_inputImages[i], inputSize * sizeof(unsigned char), hipMemcpyHostToDevice);

        tempGpuData[i] = convertToGpuData(h_gpuDataArray[i]);
        tempGpuData[i].decodedPtr = d_inputPtrs[i];
        tempGpuData[i].resizedPtr = d_outputPtrs[i];
    }

    hipMemcpy(d_gpuDataArray, tempGpuData.data(), batchSize * sizeof(GpuData), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((outSize + threads.x - 1) / threads.x,
                (outSize + threads.y - 1) / threads.y,
                batchSize);

    batchBicubicResizeKernel<<<blocks, threads>>>(d_gpuDataArray, outSize);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        for (int i = 0; i < batchSize; i++) {
            hipFree(d_inputPtrs[i]);
            hipFree(d_outputPtrs[i]);
        }
        hipFree(d_gpuDataArray);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    for (int i = 0; i < batchSize; i++) {
        hipMemcpy(h_outputImages[i], d_outputPtrs[i], outSize * outSize * sizeof(float), hipMemcpyDeviceToHost);
    }

    for (int i = 0; i < batchSize; i++) {
        hipFree(d_inputPtrs[i]);
        hipFree(d_outputPtrs[i]);
    }
    hipFree(d_gpuDataArray);
}

// Wrapper function for testing medianThresholdKernel
void testMedianThresholdKernel(
    const float** h_imgs,
    int cropSize,
    int stride,
    TestpHash* h_outHashes,
    int batchSize
) {
    float** d_imgs;
    hipMalloc(&d_imgs, batchSize * sizeof(float*));

    std::vector<float*> d_imgPtrs(batchSize);
    for (int i = 0; i < batchSize; i++) {
        hipMalloc(&d_imgPtrs[i], stride * cropSize * sizeof(float));
        hipMemcpy(d_imgPtrs[i], h_imgs[i], stride * cropSize * sizeof(float), hipMemcpyHostToDevice);
    }
    hipMemcpy(d_imgs, d_imgPtrs.data(), batchSize * sizeof(float*), hipMemcpyHostToDevice);

    pHash* d_outHashes;
    hipMalloc(&d_outHashes, batchSize * sizeof(pHash));

    int threadsPerBlock = 256; // Should be at least cropSize*cropSize
    int sharedMemSize = threadsPerBlock * sizeof(float);

    medianThresholdKernel<<<batchSize, threadsPerBlock, sharedMemSize>>>(
        (const float* const*)d_imgs, cropSize, stride, d_outHashes, batchSize
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        for (int i = 0; i < batchSize; i++) hipFree(d_imgPtrs[i]);
        hipFree(d_imgs);
        hipFree(d_outHashes);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_outHashes, d_outHashes, batchSize * sizeof(pHash), hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++) hipFree(d_imgPtrs[i]);
    hipFree(d_imgs);
    hipFree(d_outHashes);
}

// Wrapper function for testing computeLSHKeysKernel
void testComputeLSHKeysKernel(
    const size_t* h_offsets,
    const TestpHash* h_hashes,
    int n,
    const int* h_bitPositions,
    int bitsPerTable,
    int tableIndex,
    uint64_t* h_keys,
    int* h_idx
) {
    size_t* d_offsets;
    pHash* d_hashes;
    int* d_bitPositions;
    uint64_t* d_keys;
    int* d_idx;

    hipMalloc(&d_offsets, n * sizeof(size_t));
    hipMalloc(&d_hashes, n * sizeof(pHash));
    int bitPosSize = (tableIndex + 1) * bitsPerTable;
    hipMalloc(&d_bitPositions, bitPosSize * sizeof(int));
    hipMalloc(&d_keys, n * sizeof(uint64_t));
    hipMalloc(&d_idx, n * sizeof(int));

    hipMemcpy(d_offsets, h_offsets, n * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_hashes, h_hashes, n * sizeof(pHash), hipMemcpyHostToDevice);
    hipMemcpy(d_bitPositions, h_bitPositions, bitPosSize * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    computeLSHKeysKernel<<<blocks, threadsPerBlock>>>(
        d_offsets, d_hashes, n, d_bitPositions, bitsPerTable, tableIndex, d_keys, d_idx
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_offsets);
        hipFree(d_hashes);
        hipFree(d_bitPositions);
        hipFree(d_keys);
        hipFree(d_idx);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_keys, d_keys, n * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(h_idx, d_idx, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_offsets);
    hipFree(d_hashes);
    hipFree(d_bitPositions);
    hipFree(d_keys);
    hipFree(d_idx);
}

// Wrapper function for testing findPairsKernel
void testFindPairsKernel(
    const uint64_t* h_keys,
    const int* h_idx,
    int n,
    const size_t* h_offsets,
    const TestpHash* h_hashes,
    int threshold,
    TestEdge* h_outEdges,
    int* h_edgeCount,
    int maxEdges,
    int maxPairsPerBucket
) {
    uint64_t* d_keys;
    int* d_idx;
    size_t* d_offsets;
    pHash* d_hashes;
    Edge* d_outEdges;
    int* d_edgeCount;

    hipMalloc(&d_keys, n * sizeof(uint64_t));
    hipMalloc(&d_idx, n * sizeof(int));
    hipMalloc(&d_offsets, n * sizeof(size_t));
    hipMalloc(&d_hashes, n * sizeof(pHash));
    hipMalloc(&d_outEdges, maxEdges * sizeof(Edge));
    hipMalloc(&d_edgeCount, sizeof(int));

    hipMemcpy(d_keys, h_keys, n * sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(d_idx, h_idx, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, n * sizeof(size_t), hipMemcpyHostToDevice);
    hipMemcpy(d_hashes, h_hashes, n * sizeof(pHash), hipMemcpyHostToDevice);
    hipMemset(d_edgeCount, 0, sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    blocks = (blocks > 1024) ? 1024 : blocks; // Cap at reasonable number

    findPairsKernel<<<blocks, threadsPerBlock>>>(
        d_keys, d_idx, n, d_offsets, d_hashes, threshold,
        d_outEdges, d_edgeCount, maxEdges, maxPairsPerBucket
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_keys);
        hipFree(d_idx);
        hipFree(d_offsets);
        hipFree(d_hashes);
        hipFree(d_outEdges);
        hipFree(d_edgeCount);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_edgeCount, d_edgeCount, sizeof(int), hipMemcpyDeviceToHost);
    int actualEdges = (*h_edgeCount > maxEdges) ? maxEdges : *h_edgeCount;
    hipMemcpy(h_outEdges, d_outEdges, actualEdges * sizeof(Edge), hipMemcpyDeviceToHost);

    hipFree(d_keys);
    hipFree(d_idx);
    hipFree(d_offsets);
    hipFree(d_hashes);
    hipFree(d_outEdges);
    hipFree(d_edgeCount);
}