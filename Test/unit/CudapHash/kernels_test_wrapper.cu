#include "kernels.cuh"

#include "kernels_test_wrapper.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

// Wrapper function for testing dctMatrixKernel
void testDctMatrixKernel(float* h_T, int N, float invSqrtN, float sqrtTwoDivN) {
    float* d_T;

    hipMalloc(&d_T, N * N * sizeof(float));
    hipMemcpy(d_T, h_T, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 blocks(N);
    dim3 threads(N);
    dctMatrixKernel << <blocks, threads >> > (d_T, invSqrtN, sqrtTwoDivN);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(d_T);
        throw std::runtime_error(std::string("Kernel launch failed: ") + hipGetErrorString(err));
    }

    hipDeviceSynchronize();

    hipMemcpy(h_T, d_T, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_T);
}